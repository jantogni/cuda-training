#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>

#define N 5120
#define HIST 256

using namespace std;

void histograma_cpu(int * data, int * counter){
	int i;
	for(i=0; i<N; i++)
		counter[data[i]]++;
}

__global__ void histograma_gpu(int * data, int * counter){
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid < N)
		atomicAdd(&(counter[tid]),1);
}

void showDeviceProperties(){
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop,0);
        cout << "Max Threads: " << prop.maxThreadsPerBlock << endl;
        cout << "Max Grid Size: " << prop.maxGridSize[0] << " " << prop.maxGridSize[1] << " " << prop.maxGridSize[2] << endl;
        cout << "Max Threads Dim: " << prop.maxThreadsDim[0] << " " << prop.maxThreadsDim[1] << " " << prop.maxThreadsDim[2] << endl;
}


int main(){
	//showDeviceProperties();
	//#ifndef CUDA_NO_SM_11_ATOMIC_INTRINSICS
	//	cout << "Not using atomics" << endl;
	//#endif

	/* Variables */
	int * data;
	int * dev_data;

	int * counter;
	int * counter_cpu;
	int * dev_counter;

	int i;

	/*Inicialización de datos y copiado a Device*/
	data = (int *)malloc(N * sizeof(int));
	hipMalloc((void**)&dev_data, N * sizeof(int));

	counter = (int *)malloc(HIST * sizeof(int));
	counter_cpu = (int *)malloc(HIST * sizeof(int));
	hipMalloc((void**)&dev_counter, HIST * sizeof(int));

	for(i=0; i<N; i++)
		cin >> data[i];

	for(i=0; i<HIST; i++){
		counter[i] = 0;
		counter_cpu[i] = 0;
	}

	hipMemcpy(dev_counter, counter, HIST * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_data, data, N * sizeof(int), hipMemcpyHostToDevice);

	/* Llamado a CPU*/
	cout << "Resultados histograma" << endl;
	histograma_cpu(data, counter_cpu);

	/*Llamado a GPU*/
	histograma_gpu<<<ceil(N/512.0),512>>> (dev_data, dev_counter);
	//cudaThreadSynchronize();
        //cout << cudaGetErrorString(cudaGetLastError()) << endl;

	/*Copiando al host*/
	hipMemcpy(counter, dev_counter, HIST * sizeof(int), hipMemcpyDeviceToHost);

	/*Mostrando por pantalla*/
	//cout << "i\tGPU\tCPU" << endl;
	//for(i=0; i<HIST; i++)
	//	cout << i << "\t" << counter[i] << "\t" << counter_cpu[i] << endl;

	return 0;
}
