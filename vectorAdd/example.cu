#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>

#define N 3000

using namespace std;

__global__ void add(int *a, int *b, int *c) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i<N)
		c[i] = a[i] + b[i];
}

int main() {
	int *a,		*b,		*c;
	int *dev_a,	*dev_b,		*dev_c;
	int i;

	a = (int *)malloc(N*sizeof(int));
	b = (int *)malloc(N*sizeof(int));
	c = (int *)malloc(N*sizeof(int));

	hipMalloc( (void**)&dev_a, N * sizeof(int) );
	hipMalloc( (void**)&dev_b, N * sizeof(int) );
	hipMalloc( (void**)&dev_c, N * sizeof(int) );

	for(i=0; i<N; i++){
		a[i] = i;
		b[i] = 2*i;
	}
	
	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
	
	add<<< ceil(N/512.0),512>>>(dev_a, dev_b, dev_c);

	hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

	for(i=0; i<N; i++){
		cout << a[i] << " + " << b[i] << " = " << c[i] << endl;
	}
	
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}
