#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>

#define N 60

using namespace std;

__global__ void add(int *a, int *b, int *c) {
	int i = blockIdx.x;
	if(i<N)
		c[i] = a[i] + b[i];
}

int main() {
	hipDeviceProp_t prop;
	int a[N], b[N], c[N];	
	int *dev_a, *dev_b, *dev_c;

	hipMalloc( (void**)&dev_a, N * sizeof(int) );
	hipMalloc( (void**)&dev_b, N * sizeof(int) );
	hipMalloc( (void**)&dev_c, N * sizeof(int) );

	for( int i=0; i<N; i++){
		a[i] = i;
		b[i] = 2*i;
	}
	
	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
	
	add<<<N,1>>>(dev_a, dev_b, dev_c);

	hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);
	

	//for( int i=0; i<N; i++){
	//	cout << a[i] << " " << b[i] << " " << c[i] << endl;
	//}

	hipGetDeviceProperties(&prop,0);

	cout << prop.maxThreadsPerBlock << endl;

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}
